#include <iostream>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iomanip>

#define TILE_SIZE 32
#define TILE_SIZE_ADI 2
#define NDIM 2


// Kernel function for area calculation - Not cached yet
__global__ void area_kernel(double * pts, double * area, size_t nxp, size_t nyp) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //printf("BlockIdx.x: %d BlockIdx.y: %d ThreadIdx.x: %d ThreadIdx.y: %d, i: %d j: %d \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, i, j);
     size_t idx_p = (j * nxp + i) * NDIM;
     size_t idx_a = (j * nxp + i) * 5;

     if ( (i < nxp) && (j < nyp)) { // Make sure you are within the grid
         double x = pts[idx_p];
         double y = pts[idx_p + 1];        

         if ( i < (nxp-1) ) { // Assuming periodic in i direction
             double xr = pts[idx_p + NDIM];
             double yr = pts[idx_p + NDIM + 1];

             // Upward normal for (i to i+1 face)
             area[idx_a] = - (yr - y); // = alpha_{eta_x}
             area[idx_a + 1] = xr - x; // = alpha_{eta_y}
            //  printf(" Area - i-i+1 = %e, %e", -(yr -y), (xr - x) );

         } else {
             area[idx_a] = 0.0;
             area[idx_a + 1] = 0.0;
         }

         if ( j < (nyp -1) ) {
             double xu = pts[idx_p + nxp * NDIM];
             double yu = pts[idx_p + nxp * NDIM + 1];

             // Rightward normal for (j to j+1 face)
             area[idx_a + 2] = (yu - y); // = alpha_{xi_x}
             area[idx_a + 3] = -(xu - x); // = alpha_{xi_y}
            //  printf(" Area - i-i+1 = %e, %e", -(yu -y), (xu - x) );

         } else {
             area[idx_a + 2] = 0.0;
             area[idx_a + 3] = 0.0;
         }

         if ( (i < (nxp-1)) && (j < (nyp -1)) ) {
            
            double xr = pts[idx_p + NDIM];
            double yr = pts[idx_p + NDIM + 1];
            double xu = pts[idx_p + nxp * NDIM];
            double yu = pts[idx_p + nxp * NDIM + 1];
            double xur = pts[idx_p + nxp * NDIM + NDIM];
            double yur = pts[idx_p + nxp * NDIM + NDIM + 1];
    
            // ad = (xur - x), (yur - y)
            // bc = (xu - xr), (yu - yr)
            // Cross product - ad x bc
            area[idx_a + 4] = 0.5 * ( (xur - x) * (yu - yr) -  (xu - xr) * (yur - y) );
         }
     }
}

// Kernel to calculate cell centers
__global__ void compute_cellcenter(double * pts, double * cell_center, size_t nx, size_t ny, size_t nxp, size_t nyp) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    size_t idx_p = ( (j * nxp) + i ) * NDIM;
    size_t idx_c = ( (j * nx) + i ) * NDIM ;

    if ( (i < nx) && (j < ny)) { // Make sure you're within the grid

        double xij = pts[idx_p];
        double yij = pts[idx_p + 1];
        double xip1j = pts[idx_p + NDIM];
        double yip1j = pts[idx_p + NDIM + 1];
        double xijp1 = pts[idx_p + nxp * NDIM];
        double yijp1 = pts[idx_p + nxp * NDIM + 1];
        double xip1jp1 = pts[idx_p + nxp * NDIM + NDIM];
        double yip1jp1 = pts[idx_p + nxp * NDIM + NDIM + 1];

        double x = 0.25 * (xij + xip1j + xijp1 + xip1jp1);
        double y = 0.25 * (yij + yip1j + yijp1 + yip1jp1);

        cell_center[idx_c] = x;
        cell_center[idx_c + 1] = y;
    }

}

// Kernel to initialize a field phi as x^2 + y^3
__global__ void initialize_phi(double * pts, double * phi, double * phi_bc_bot, double * phi_bc_top, size_t nx, size_t ny, size_t nxp, size_t nyp) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    size_t idx_p = ( (j * nxp) + i ) * NDIM;
    size_t idx_phi = ( (j * nx) + i ) ;

    if ( (i < nx) && (j < ny)) { // Make sure you're within the grid

        double xij = pts[idx_p];
        double yij = pts[idx_p + 1];
        double xip1j = pts[idx_p + NDIM];
        double yip1j = pts[idx_p + NDIM + 1];
        double xijp1 = pts[idx_p + nxp * NDIM];
        double yijp1 = pts[idx_p + nxp * NDIM + 1];
        double xip1jp1 = pts[idx_p + nxp * NDIM + NDIM];
        double yip1jp1 = pts[idx_p + nxp * NDIM + NDIM + 1];

        double x = 0.25 * (xij + xip1j + xijp1 + xip1jp1);
        double y = 0.25 * (yij + yip1j + yijp1 + yip1jp1);

        phi[idx_phi] = x * x + y * y * y;

        if (j == 0) {
            x = 0.5 * (xij + xip1j);
            y = 0.5 * (yij + yip1j);
            phi_bc_bot[i] = x * x + y * y * y;
        } else if ( j == (ny - 1)) {
            x = 0.5 * (xijp1 + xip1jp1);
            y = 0.5 * (yijp1 + yip1jp1);
            phi_bc_top[i] = x * x + y * y * y;
        }

    }

}

// Kernel to compute gradient of the reference field phi as 2x iHat + 3y^2 jHat
__global__ void reference_grad_phi(double * pts, double * grad_phi_ref, size_t nx, size_t ny, size_t nxp, size_t nyp) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    size_t idx_p = ( (j * nxp) + i ) * NDIM;
    size_t idx_grad_phi = ( (j * nx) + i ) * NDIM;

    if ( (i < nx) && (j < ny)) { // Make sure you're within the grid

        double xij = pts[idx_p];
        double yij = pts[idx_p + 1];
        double xip1j = pts[idx_p + NDIM];
        double yip1j = pts[idx_p + NDIM + 1];
        double xijp1 = pts[idx_p + nxp * NDIM];
        double yijp1 = pts[idx_p + nxp * NDIM + 1];
        double xip1jp1 = pts[idx_p + nxp * NDIM + NDIM];
        double yip1jp1 = pts[idx_p + nxp * NDIM + NDIM + 1];

        double x = 0.25 * (xij + xip1j + xijp1 + xip1jp1);
        double y = 0.25 * (yij + yip1j + yijp1 + yip1jp1);

        grad_phi_ref[idx_grad_phi] = 2.0 * x;
        grad_phi_ref[idx_grad_phi + 1] = 3.0 * y * y;

    }

}

// Kernel to compute vector gradient of phi using Gauss theorem
/*
   
*/
__global__ void vector_grad_gauss(double * phi, double * grad_phi, double * area, double * phi_bc_bot, double * phi_bc_top, size_t nx, size_t ny, size_t nxp, size_t nyp) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    size_t idx_phi = (j * nx + i);
    size_t idx_phi_ip1 = (j * nx + (i+1)%nx );
    size_t idx_phi_im1 = (j * nx + (i-1)*nx );
    size_t idx_gp = (j * nx + i) * NDIM;

    size_t idx_a = (j * nxp + i) * 5;



    if ( (i < nx) && (j < ny)) { // Make sure you're within the grid
        printf("BlockIdx.x %d, BlockIdx.y %d, threadIdx.x %d, threadIdx.y %d, i = %d, j = %d - idx_gp %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, i, j, idx_gp);
        double phi_xix_e = 0.0;
        double phi_xix_w = 0.0;
        double phi_xiy_e = 0.0;
        double phi_xiy_w = 0.0;

        double phi_etax_n = 0.0;
        double phi_etay_n = 0.0;
        double phi_etax_s = 0.0;
        double phi_etay_s = 0.0;

        double phiij = phi[idx_phi];

        if ( j == 0) {

            double phiijp1 = phi[idx_phi + nx];
            phi_etax_s = phi_bc_bot[i]  * area[idx_a];
            phi_etax_n = 0.5 * (phiijp1 + phiij) * area[idx_a + nxp * 5];
            phi_etay_s = phi_bc_bot[i]  * area[idx_a + 1];
            phi_etay_n = 0.5 * (phiijp1 + phiij) * area[idx_a + nxp * 5 + 1];         

        } else if (j == (ny - 1)) {

            double phiijm1 = phi[idx_phi - nx];
            phi_etax_s = 0.5 * (phiijm1 + phiij) * area[idx_a];
            phi_etax_n = phi_bc_top[i]  * area[idx_a + nxp * 5];
            phi_etay_s = 0.5 * (phiijm1 + phiij) * area[idx_a + 1];
            phi_etay_n = phi_bc_top[i]  * area[idx_a + nxp * 5 + 1];         

        } else {

            double phiijm1 = phi[idx_phi - nx];
            double phiijp1 = phi[idx_phi + nx];
            phi_etax_s = 0.5 * (phiijm1 + phiij) * area[idx_a];
            phi_etax_n = 0.5 * (phiijp1 + phiij) * area[idx_a + nxp * 5];
            phi_etay_s = 0.5 * (phiijm1 + phiij) * area[idx_a + 1];
            phi_etay_n = 0.5 * (phiijp1 + phiij) * area[idx_a + nxp * 5 + 1];         

        }
    
        double phiip1j = phi[idx_phi_ip1];
        double phiim1j = phi[idx_phi_im1];
        phi_xix_e = 0.5 * ( phiip1j + phiij) * area[idx_a + 5 + 2];
        phi_xix_w = 0.5 * ( phiim1j + phiij) * area[idx_a + 2];
        phi_xiy_e = 0.5 * ( phiip1j + phiij) * area[idx_a + 5 + 3];
        phi_xiy_w = 0.5 * ( phiim1j + phiij) * area[idx_a + 3];
        
        // printf("i %d j %d - %d \n", i, j, idx_gp);
        grad_phi[idx_gp] = (phi_xix_e - phi_xix_w + phi_etax_n - phi_etax_s)/area[idx_a + 4];
        grad_phi[idx_gp + 1] = (phi_xiy_e - phi_xiy_w + phi_etay_n - phi_etay_s)/area[idx_a + 4];

    }
}


int main() {

    // Read the airfoil data

    std::ifstream plot3dfile("du00w212.x");

    if (!plot3dfile.is_open()) {
        std::cerr << "Error: could not open file" << std::endl;
        return 1;
    }

    size_t nxp, nyp, ntotp;
    plot3dfile >> nxp >> nyp;
    std::cout << "nxp: " << nxp << " nyp: " << nyp << std::endl;
    ntotp = nxp * nyp ;
    double * h_pts = new double[ntotp * NDIM];
    
    for (size_t idim = 0; idim < NDIM; idim++) {
        for (size_t i = 0; i < ntotp; i++) {
            plot3dfile >> h_pts[NDIM * i + idim];
        }
    }

    plot3dfile.close();

    // Allocate memory on the device
    double * pts;
    hipMalloc(&pts, ntotp * NDIM * sizeof(double));
    hipMemcpy(pts, h_pts, ntotp * NDIM * sizeof(double), hipMemcpyHostToDevice);

    double * area;
    hipMalloc(&area, ntotp * 5 * sizeof(double));


    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    dim3 grid_p((nxp + block.x - 1) / block.x, (nyp + block.y - 1) / block.y, 1);

    printf("Grid: %d %d Block: %d %d\n", grid_p.x, grid_p.y, block.x, block.y);
    area_kernel<<<grid_p, block>>>(pts, area, nxp, nyp);

    
    // Quantities defined at cell centers
    size_t nx = nxp - 1;
    size_t ny = nyp - 1;
    size_t ntot = nx * ny;
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, 1);
    
    // Create a field phi and initialize it
    double * phi;
    hipMalloc(&phi, ntot * sizeof(double));
    double * phi_bc_bot;
    double * phi_bc_top;
    hipMalloc(&phi_bc_bot, nx * sizeof(double));
    hipMalloc(&phi_bc_top, nx * sizeof(double));

    double * cell_center;
    hipMalloc(&cell_center, ntot * NDIM * sizeof(double));
    compute_cellcenter<<<grid, block>>>(pts, cell_center, nx, ny, nxp, nyp);

    initialize_phi<<<grid, block>>>(pts, phi, phi_bc_bot, phi_bc_top, nx, ny, nxp, nyp);
    hipDeviceSynchronize();

    double * h_phi = new double [ntot];
    hipMemcpy(h_phi, phi, ntot * sizeof(double), hipMemcpyDeviceToHost);
    
    double * grad_phi_ref;
    hipMalloc(&grad_phi_ref, ntot * NDIM * sizeof(double));
    reference_grad_phi<<<grid, block>>>(pts, grad_phi_ref, nx, ny, nxp, nyp);
    hipDeviceSynchronize();
    double * h_grad_phi_ref = new double[ntot * NDIM];
    hipMemcpy(h_grad_phi_ref, grad_phi_ref, ntot * NDIM * sizeof(double), hipMemcpyDeviceToHost);

    double * grad_phi;
    hipMalloc(&grad_phi, ntot * NDIM * sizeof(double));
    // Calculate the gradient of phi
    vector_grad_gauss<<<grid, block>>>(phi, grad_phi, area, phi_bc_bot, phi_bc_top, nx, ny, nxp, nyp);
    hipDeviceSynchronize();
 
    // Check for launch errors
    hipError_t err = hipGetLastError();
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;

    double * h_grad_phi = new double[ntot * NDIM];
    hipMemcpy(h_grad_phi, grad_phi, ntot * NDIM * sizeof(double), hipMemcpyDeviceToHost);

    // Write h_grad_phi to a file
    double zero = 0.0;
    std::ofstream grad_file("grad_phi.vtk");
    if (grad_file.is_open()) {
        grad_file << "# vtk DataFile Version 3.0" << std::endl;
        grad_file << "DU00W2121 Airfoil " << std::endl;
        grad_file << "ASCII" << std::endl;
        grad_file << "DATASET STRUCTURED_GRID " << std::endl;
        grad_file << "DIMENSIONS " << nxp << " " << nyp << " 1" << std::endl;
        grad_file << "POINTS " << nxp * nyp * 1 << " double" << std::endl;
        for (size_t i = 0; i < ntotp; ++i) 
            grad_file << std::fixed << std::setprecision(6) <<h_pts[i * NDIM] << " " << h_pts[i * NDIM + 1] << " " << zero << std::endl;
        grad_file << "CELL_DATA " << nx * ny << std::endl;
        grad_file << "SCALARS phi double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (size_t i = 0; i < ntot; ++i) 
            grad_file << h_phi[i] << std::endl;
        grad_file << "SCALARS grad_phi_x double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (size_t i = 0; i < ntot; ++i) 
            grad_file << h_grad_phi[i * NDIM] << std::endl;
        grad_file << "SCALARS grad_phi_y double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (size_t i = 0; i < ntot; ++i) 
            grad_file << h_grad_phi[i * NDIM + 1] << std::endl;
        grad_file << "SCALARS grad_phi_x_ref double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (size_t i = 0; i < ntot; ++i) 
            grad_file << h_grad_phi_ref[i * NDIM] << std::endl;
        grad_file << "SCALARS grad_phi_y_ref double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (size_t i = 0; i < ntot; ++i) 
            grad_file << h_grad_phi_ref[i * NDIM + 1] << std::endl;
        grad_file.close();
        std::cout << "Gradient of phi written to grad_phi.vtk" << std::endl;
    } else {
        std::cerr << "Error: could not open grad_phi.vtk for writing" << std::endl;
    }

    delete [] h_pts;
    hipFree(pts);
    hipFree(area);
    hipFree(phi);
    delete [] h_phi;
    
    // cudaFree(grad_phi);
    delete [] h_grad_phi;
    // cudaFree(grad_phi_ref);
    delete [] h_grad_phi_ref;
    
    return 0;
}