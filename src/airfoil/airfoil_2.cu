#include <iostream>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <iomanip>

#define TILE_SIZE 16
#define TILE_SIZE_ADI 2
#define NDIM 2


// Kernel function for area calculation - Not cached yet
__global__ void area_kernel(double * pts, double * area, int nxp, int nyp) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    //printf("BlockIdx.x: %d BlockIdx.y: %d ThreadIdx.x: %d ThreadIdx.y: %d, i: %d j: %d \n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, i, j);
     int idx_p = (j * nxp + i) * NDIM;
     int idx_a = (j * nxp + i) * 7;

     if ( (i < nxp) && (j < nyp)) { // Make sure you are within the grid
         double x = pts[idx_p];
         double y = pts[idx_p + 1];        

         if ( i < (nxp-1) ) { // Assuming periodic in i direction
             double xr = pts[idx_p + NDIM];
             double yr = pts[idx_p + NDIM + 1];

             // Upward normal for (i to i+1 face)
             area[idx_a] = - (yr - y); // = alpha_{eta_x}
             area[idx_a + 1] = xr - x; // = alpha_{eta_y}
            //  printf(" Area - i-i+1 = %e, %e", -(yr -y), (xr - x) );

         } else {
             area[idx_a] = 0.0;
             area[idx_a + 1] = 0.0;
         }

         if ( j < (nyp -1) ) {
             double xu = pts[idx_p + nxp * NDIM];
             double yu = pts[idx_p + nxp * NDIM + 1];

             // Rightward normal for (j to j+1 face)
             area[idx_a + 2] = (yu - y); // = alpha_{xi_x}
             area[idx_a + 3] = -(xu - x); // = alpha_{xi_y}
            //  printf(" Area - i-i+1 = %e, %e", -(yu -y), (xu - x) );

         } else {
             area[idx_a + 2] = 0.0;
             area[idx_a + 3] = 0.0;
         }

         if ( (i < (nxp-1)) && (j < (nyp -1)) ) {
            
            double xr = pts[idx_p + NDIM];
            double yr = pts[idx_p + NDIM + 1];
            double xu = pts[idx_p + nxp * NDIM];
            double yu = pts[idx_p + nxp * NDIM + 1];
            double xur = pts[idx_p + nxp * NDIM + NDIM];
            double yur = pts[idx_p + nxp * NDIM + NDIM + 1];
    
            // ad = (xur - x), (yur - y)
            // bc = (xu - xr), (yu - yr)
            // Cross product - ad x bc
            area[idx_a + 4] = 0.5 * ( (xur - x) * (yu - yr) -  (xu - xr) * (yur - y) );
         }
     }
}


// Kernel function for interpolating factor calculation - Not cached yet
/*

    Method to find intersection point between two lines. 

    Line 1 from p1 to p2. Line 2 from p3 to p4
    Any point from p1 to p2 = p1 + t1 * (p2 - p1)
    Any point from p3 to p4 = p3 + t2 * (p4 - p3)
    We want these points to be the same. Hence, we solve equation system formed by equating these two coordinates in x and y.
    p1 + t1 * (p2 - p1) = p3 + t2 * (p4 - p3)
    
    (p2.x - p1.x) * t1 + (p3.x - p4.x) * t2 = p3.x - p1.x
    (p2.y - p1.y) * t1 + (p3.y - p4.y) * t2 = p3.y - p1.y

    detA = (p2.x - p1.x) * (p3.y - p4.y) - (p3.x - p4.x) * (p2.y - p1.y)

    t1 = ((p3.y - p4.y) * (p3.x - p1.x) - (p3.x - p4.x) * (p3.y - p1.y)) / detA
    t2 = (-(p2.y - p1.y) * (p3.x - p1.x) + (p2.x - p1.x) * (p3.y - p1.y)) / detA

)
*/
__global__ void compute_if(double * pts, double * cell_center, double * area, int nx, int ny, int nxp, int nyp) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int idx_p = (j * nxp + i) * NDIM;
    int idx_a = (j * nxp + i) * 7;
    int idx_cc = (j * nx + i%nx) * NDIM;

    if ( (i < nxp) && (j < ny)) { // Make sure you're within the grid
     
    
        //p3
        double x = pts[idx_p];
        double y = pts[idx_p + 1];        
        //p4
        double xu = pts[idx_p + nxp * NDIM] ;
        double yu = pts[idx_p + nxp * NDIM + 1];

        //p1
        double xim1 = 0.0;
        double yim1 = 0.0;
        //p2
        double xi = 0.0;
        double yi = 0.0;
        //p1
        if (i == 0) {
            xim1 = cell_center[idx_cc + (nx-1)*NDIM];
            yim1 = cell_center[idx_cc + (nx-1)*NDIM + 1];
            xi = cell_center[idx_cc];
            yi = cell_center[idx_cc+1];           
        } else if (i == (nxp-1)) {
            xim1 = cell_center[j * nx + (nx-1)*NDIM];
            yim1 = cell_center[j * nx + (nx-1)*NDIM+1];
            xi = cell_center[j * nx];
            yi = cell_center[j * nx + 1];
        } else {
            xim1 = cell_center[idx_cc-NDIM];
            yim1 = cell_center[idx_cc-NDIM+1];
            xi = cell_center[idx_cc];
            yi = cell_center[idx_cc+1];
        }

        double detA = (xi-xim1) * (y-yu) - (x-xu) * (yi-yim1);
        double t1 = ( (y-yu) * (x-xim1) - (x-xu) * (y-yim1) ) / detA ;

        if ( std::isinf(t1) || std::isnan(t1)) {
            printf("i %d, j %d - t1 = %e, x, y = (%e, %e), xu, yu = (%e, %e), xim1, yim1 = (%e, %e), xi, yi = (%e, %e) \n", i , j, t1, x, y, xu, yu, xim1, yim1, xi, yi );
        }
        area[idx_a + 5] = t1;

    if ( (j > 0) && (j < ny) ) {
        //p3
        double x = pts[idx_p];
        double y = pts[idx_p + 1];        
        //p4
        double xr = pts[idx_p + NDIM];
        double yr = pts[idx_p + NDIM + 1];

        //p1
        double xjm1 = cell_center[idx_cc-nx*NDIM];
        double yjm1 = cell_center[idx_cc-nx*NDIM+1];        
        //p2
        double xj = cell_center[idx_cc];
        double yj = cell_center[idx_cc+1];


    /*
    detA = (p2.x - p1.x) * (p3.y - p4.y) - (p3.x - p4.x) * (p2.y - p1.y)

    t1 = ((p3.y - p4.y) * (p3.x - p1.x) - (p3.x - p4.x) * (p3.y - p1.y)) / detA
    */        
        double detA = (xj - xjm1) * (y - yr) - (x - xr) * (yj - yjm1) ;
        double t1 = ( (y - yr) * (x - xjm1) - (x - xr) * (y - yjm1) ) / detA ;

        if (std::isinf(t1) || std::isnan(t1)) {
            printf( "i %d, j %d - t1 = %e, x, y = (%e, %e), xr, yr = (%e, %e), xjm1, yjm1 = (%e, %e), xj, yj = (%e, %e) \n", i , j, t1, x, y, xr, yr, xjm1, yjm1, xj, yj);
        }

        area[idx_a + 6] = t1;
    } else {
        area[idx_a + 6] = 0.0;
    }

    // if (j > 90) 
    //     printf("i: %d j: %d, if_y: %e, if_x: %e \n", i, j, area[idx_a + 5], area[idx_a + 6]);
    
    }



}

// Kernel to calculate cell centers
__global__ void compute_cellcenter(double * pts, double * cell_center, int nx, int ny, int nxp, int nyp) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    int idx_p = ( (j * nxp) + i ) * NDIM;
    int idx_c = ( (j * nx) + i ) * NDIM ;

    if ( (i < nx) && (j < ny)) { // Make sure you're within the grid

        double xij = pts[idx_p];
        double yij = pts[idx_p + 1];
        double xip1j = pts[idx_p + NDIM];
        double yip1j = pts[idx_p + NDIM + 1];
        double xijp1 = pts[idx_p + nxp * NDIM];
        double yijp1 = pts[idx_p + nxp * NDIM + 1];
        double xip1jp1 = pts[idx_p + nxp * NDIM + NDIM];
        double yip1jp1 = pts[idx_p + nxp * NDIM + NDIM + 1];

        double x = 0.25 * (xij + xip1j + xijp1 + xip1jp1);
        double y = 0.25 * (yij + yip1j + yijp1 + yip1jp1);

        cell_center[idx_c] = x;
        cell_center[idx_c + 1] = y;
    }

}

// Kernel to initialize a field phi as x^2 + y^3
__global__ void initialize_phi(double * pts, double * phi, double * phi_bc_bot, double * phi_bc_top, int nx, int ny, int nxp, int nyp) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    int idx_p = ( (j * nxp) + i ) * NDIM;
    int idx_phi = ( (j * nx) + i ) ;

    if ( (i < nx) && (j < ny)) { // Make sure you're within the grid

        double xij = pts[idx_p];
        double yij = pts[idx_p + 1];
        double xip1j = pts[idx_p + NDIM];
        double yip1j = pts[idx_p + NDIM + 1];
        double xijp1 = pts[idx_p + nxp * NDIM];
        double yijp1 = pts[idx_p + nxp * NDIM + 1];
        double xip1jp1 = pts[idx_p + nxp * NDIM + NDIM];
        double yip1jp1 = pts[idx_p + nxp * NDIM + NDIM + 1];

        double x = 0.25 * (xij + xip1j + xijp1 + xip1jp1);
        double y = 0.25 * (yij + yip1j + yijp1 + yip1jp1);

        phi[idx_phi] = x * x + y * y * y;

        if (j == 0) {
            x = 0.5 * (xij + xip1j);
            y = 0.5 * (yij + yip1j);
            phi_bc_bot[i] = x * x + y * y * y;
        } else if ( j == (ny - 1)) {
            x = 0.5 * (xijp1 + xip1jp1);
            y = 0.5 * (yijp1 + yip1jp1);
            phi_bc_top[i] = x * x + y * y * y;
        }

    }

}

// Kernel to compute gradient of the reference field phi as 2x iHat + 3y^2 jHat
__global__ void reference_grad_phi(double * pts, double * grad_phi_ref, int nx, int ny, int nxp, int nyp) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    int idx_p = ( (j * nxp) + i ) * NDIM;
    int idx_grad_phi = ( (j * nx) + i ) * NDIM;

    if ( (i < nx) && (j < ny)) { // Make sure you're within the grid

        double xij = pts[idx_p];
        double yij = pts[idx_p + 1];
        double xip1j = pts[idx_p + NDIM];
        double yip1j = pts[idx_p + NDIM + 1];
        double xijp1 = pts[idx_p + nxp * NDIM];
        double yijp1 = pts[idx_p + nxp * NDIM + 1];
        double xip1jp1 = pts[idx_p + nxp * NDIM + NDIM];
        double yip1jp1 = pts[idx_p + nxp * NDIM + NDIM + 1];

        double x = 0.25 * (xij + xip1j + xijp1 + xip1jp1);
        double y = 0.25 * (yij + yip1j + yijp1 + yip1jp1);

        grad_phi_ref[idx_grad_phi] = 2.0 * x;
        grad_phi_ref[idx_grad_phi + 1] = 3.0 * y * y;

    }

}

// Kernel to compute vector gradient of phi
/*

    There are 5 quantities for every point.
    1. Jacobian ( x_psi * y_eta - y_psi * x_eta)
    2. alpha_psi_x = y_eta
    3. alpha_psi_y = -x_eta
    4. alpha_eta_x = y_psi
    5. alpha_eta_y = x_psi

    Once these quantities are computed, the x- and y-derivative of a variable phi are
    
        phi_x = 1/J ( (phi * alpha_psi_x)_psi + (phi * alpha_eta_x)_eta )
        phi_y = 1/J ( (phi * alpha_psi_y)_psi + (phi * alpha_eta_y)_eta )
   
*/
__global__ void vector_grad_gauss(double * phi, double * grad_phi, double * grad_phi_ref,  double * area, double * phi_bc_bot, double * phi_bc_top, int nx, int ny, int nxp, int nyp) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int idx_phi = (j * nx + i);
    int idx_phi_ip1 = (j * nx + (i+1)%nx );
    int idx_phi_im1 = (j * nx + (i-1) );
    if (i == 0)
        idx_phi_im1 = (j * nx + nx-1 );
    int idx_gp = (j * nx + i) * NDIM;

    int idx_a = (j * nxp + i) * 7;



    if ( (i < nx) && (j < ny)) { // Make sure you're within the grid
        // printf("BlockIdx.x %d, BlockIdx.y %d, threadIdx.x %d, threadIdx.y %d, i = %d, j = %d - idx_gp %d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, i, j, idx_gp);
        double phi_xix_e = 0.0;
        double phi_xix_w = 0.0;
        double phi_xiy_e = 0.0;
        double phi_xiy_w = 0.0;

        double phi_etax_n = 0.0;
        double phi_etay_n = 0.0;
        double phi_etax_s = 0.0;
        double phi_etay_s = 0.0;

        double phiij = phi[idx_phi];
        double phiijp1 = 0.0;
        double phiijm1 = 0.0;

        if (j < (ny -1 ))
            phiijp1 = phi[idx_phi + nx];
        if (j > 0)
            phiijm1 = phi[idx_phi - nx];

        if ( j == 0) {

            double phiijp1 = phi[idx_phi + nx];
            phi_etax_s = phi_bc_bot[i]  * area[idx_a];
            phi_etax_n = (area[idx_a + nxp * 7 + 5] * phiijp1 + (1.0 - area[idx_a + nxp * 7 + 5]) * phiij ) * area[idx_a + nxp * 7];
            phi_etay_s = phi_bc_bot[i]  * area[idx_a + 1];
            phi_etay_n = (area[idx_a + nxp * 7 + 5] * phiijp1 + (1.0 - area[idx_a + nxp * 7 + 5]) * phiij ) * area[idx_a + nxp * 7 + 1];         

        } else if (j == (ny - 1)) {

            double phiijm1 = phi[idx_phi - nx];
            phi_etax_s =  ( area[idx_a + 5] * phiij + (1.0 - area[idx_a + 5]) * phiijm1 ) * area[idx_a];
            phi_etax_n = phi_bc_top[i]  * area[idx_a + nxp * 7];
            phi_etay_s = ( area[idx_a + 5] * phiij + (1.0 - area[idx_a + 5]) * phiijm1 ) * area[idx_a + 1];
            phi_etay_n = phi_bc_top[i]  * area[idx_a + nxp * 7 + 1];         

        } else {

            double phiijm1 = phi[idx_phi - nx];
            double phiijp1 = phi[idx_phi + nx];
            phi_etax_s = (area[idx_a + 5] * phiij + (1.0 - area[idx_a + 5]) * phiijm1 ) * area[idx_a];
            phi_etax_n = (area[idx_a + nxp * 7 + 5] * phiijp1 + (1.0 - area[idx_a + nxp * 7 + 5]) * phiij ) * area[idx_a + nxp * 7];
            phi_etay_s = (area[idx_a + 5] * phiij + (1.0 - area[idx_a + 5]) * phiijm1 ) * area[idx_a + 1];
            phi_etay_n = (area[idx_a + nxp * 7 + 5] * phiijp1 + (1.0 - area[idx_a + nxp * 7 + 5]) * phiij ) * area[idx_a + nxp * 7 + 1];         

        }
    
        double phiip1j = phi[idx_phi_ip1];
        double phiim1j = phi[idx_phi_im1];
        phi_xix_w = ( area[idx_a + 6] * phiij + (1.0 - area[idx_a + 6]) * phiim1j ) * area[idx_a + 2];
        phi_xix_e = ( area[idx_a + 7 + 6] * phiip1j + (1.0 - area[idx_a + 7 + 6]) * phiij ) * area[idx_a + 7 + 2];
        phi_xiy_w = ( area[idx_a + 6] * phiij + (1.0 - area[idx_a + 6]) * phiim1j ) * area[idx_a + 3];
        phi_xiy_e = ( area[idx_a + 7 + 6] * phiip1j + (1.0 - area[idx_a + 7 + 6]) * phiij ) * area[idx_a + 7 + 3];
        
        // printf("i %d j %d - %d \n", i, j, idx_gp);
        double tmp = (phi_xix_e - phi_xix_w + phi_etax_n - phi_etax_s)/area[idx_a + 4];
        double tmp1 = (phi_xiy_e - phi_xiy_w + phi_etay_n - phi_etay_s)/area[idx_a + 4];
        grad_phi[idx_gp] = tmp;
        grad_phi[idx_gp + 1] = tmp1;

        if ( (tmp * grad_phi_ref[idx_gp]) < 0.0 )
            printf("i %d, j %d, grad_phi_x = %e, grad_phi_ref_x %e, phi_x_ew = %e, phi_x_ns = %e, IF: (S) %e, (W) %e, (N) %e, (E) %e, Total %e, Phi (I-1) %e, (I), %e, (I+1) %e, (J-1) %e, (J+1) %e \n", i, j, tmp, grad_phi_ref[idx_gp], (phi_xix_e - phi_xix_w)/area[idx_a+4], (phi_etax_n - phi_etax_s)/area[idx_a+4], area[idx_a+5], area[idx_a+6], area[idx_a+nxp*7+5], area[idx_a+7+5], area[idx_a+4], phiim1j, phiij, phiip1j, phiijm1, phiijp1);
        // if (i == 0)
        //     printf("i %d, j %d, grad_phi_x = %e, phi_xiy_e = %e, phi_xiy_w = %e, phi_etay_n = %e, phi_etay_s = %e, Areas: (S) %e, %e, (N) %e, %e, (E) %e, %e, (W) %e, %e, Phi (I-1) %e, (I), %e, (I+1) %e, (J-1) %e, (J+1) %e\n", i, j, tmp1, phi_xiy_e, phi_xiy_w, phi_etay_n, phi_etay_s, area[idx_a], area[idx_a+1], area[idx_a+nxp*7], area[idx_a+nxp*7+1], area[idx_a+7+2], area[idx_a+7+3], area[idx_a+2], area[idx_a+3], phiim1j, phiij, phiip1j, phiijm1, phiijp1);
    }
}

int main() {

    // Read the airfoil data

    std::ifstream plot3dfile("du00w212.x");

    if (!plot3dfile.is_open()) {
        std::cerr << "Error: could not open file" << std::endl;
        return 1;
    }

    int nxp, nyp, ntotp;
    plot3dfile >> nxp >> nyp;
    std::cout << "nxp: " << nxp << " nyp: " << nyp << std::endl;
    ntotp = nxp * nyp ;
    double * h_pts = new double[ntotp * NDIM];
    
    for (int idim = 0; idim < NDIM; idim++) {
        for (int i = 0; i < ntotp; i++) {
            plot3dfile >> h_pts[NDIM * i + idim];
        }
    }

    plot3dfile.close();

    // Allocate memory on the device
    double * pts;
    hipMalloc(&pts, ntotp * NDIM * sizeof(double));
    hipMemcpy(pts, h_pts, ntotp * NDIM * sizeof(double), hipMemcpyHostToDevice);

    double * area;
    hipMalloc(&area, ntotp * 7 * sizeof(double));


    dim3 block(TILE_SIZE, TILE_SIZE, 1);
    dim3 grid_p((nxp + block.x - 1) / block.x, (nyp + block.y - 1) / block.y, 1);

    printf("Grid: %d %d Block: %d %d\n", grid_p.x, grid_p.y, block.x, block.y);
    area_kernel<<<grid_p, block>>>(pts, area, nxp, nyp);

    
    // Quantities defined at cell centers
    int nx = nxp - 1;
    int ny = nyp - 1;
    int ntot = nx * ny;
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y, 1);
    
    // Create a field phi and initialize it
    double * phi;
    hipMalloc(&phi, ntot * sizeof(double));
    double * phi_bc_bot;
    double * phi_bc_top;
    hipMalloc(&phi_bc_bot, nx * sizeof(double));
    hipMalloc(&phi_bc_top, nx * sizeof(double));

    double * cell_center;
    hipMalloc(&cell_center, ntot * NDIM * sizeof(double));
    compute_cellcenter<<<grid, block>>>(pts, cell_center, nx, ny, nxp, nyp);
    hipDeviceSynchronize();
    compute_if<<<grid, block>>>(pts, cell_center, area, nx, ny, nxp, nyp);
    hipDeviceSynchronize();

    initialize_phi<<<grid, block>>>(pts, phi, phi_bc_bot, phi_bc_top, nx, ny, nxp, nyp);
    hipDeviceSynchronize();

    double * h_phi = new double [ntot];
    hipMemcpy(h_phi, phi, ntot * sizeof(double), hipMemcpyDeviceToHost);
    
    double * grad_phi_ref;
    hipMalloc(&grad_phi_ref, ntot * NDIM * sizeof(double));
    reference_grad_phi<<<grid, block>>>(pts, grad_phi_ref, nx, ny, nxp, nyp);
    hipDeviceSynchronize();
    double * h_grad_phi_ref = new double[ntot * NDIM];
    hipMemcpy(h_grad_phi_ref, grad_phi_ref, ntot * NDIM * sizeof(double), hipMemcpyDeviceToHost);

    double * grad_phi;
    hipMalloc(&grad_phi, ntot * NDIM * sizeof(double));
    // Calculate the gradient of phi
    vector_grad_gauss<<<grid, block>>>(phi, grad_phi, grad_phi_ref, area, phi_bc_bot, phi_bc_top, nx, ny, nxp, nyp);
    hipDeviceSynchronize();
 
    // Check for launch errors
    hipError_t err = hipGetLastError();
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;

    double * h_grad_phi = new double[ntot * NDIM];
    hipMemcpy(h_grad_phi, grad_phi, ntot * NDIM * sizeof(double), hipMemcpyDeviceToHost);

    // Write h_grad_phi to a file
    double zero = 0.0;
    std::ofstream grad_file("grad_phi.vtk");
    if (grad_file.is_open()) {
        grad_file << "# vtk DataFile Version 3.0" << std::endl;
        grad_file << "DU00W2121 Airfoil " << std::endl;
        grad_file << "ASCII" << std::endl;
        grad_file << "DATASET STRUCTURED_GRID " << std::endl;
        grad_file << "DIMENSIONS " << nxp << " " << nyp << " 1" << std::endl;
        grad_file << "POINTS " << nxp * nyp * 1 << " double" << std::endl;
        for (int i = 0; i < ntotp; ++i) 
            grad_file << std::fixed << std::setprecision(6) <<h_pts[i * NDIM] << " " << h_pts[i * NDIM + 1] << " " << zero << std::endl;
        grad_file << "CELL_DATA " << nx * ny << std::endl;
        grad_file << "SCALARS phi double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (int i = 0; i < ntot; ++i) 
            grad_file << h_phi[i] << std::endl;
        grad_file << "SCALARS grad_phi_x double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (int i = 0; i < ntot; ++i) 
            grad_file << h_grad_phi[i * NDIM] << std::endl;
        grad_file << "SCALARS grad_phi_y double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (int i = 0; i < ntot; ++i) 
            grad_file << h_grad_phi[i * NDIM + 1] << std::endl;
        grad_file << "SCALARS grad_phi_x_ref double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (int i = 0; i < ntot; ++i) 
            grad_file << h_grad_phi_ref[i * NDIM] << std::endl;
        grad_file << "SCALARS grad_phi_y_ref double 1" << std::endl;
        grad_file << "LOOKUP_TABLE default" << std::endl;
        for (int i = 0; i < ntot; ++i) 
            grad_file << h_grad_phi_ref[i * NDIM + 1] << std::endl;
        grad_file.close();
        std::cout << "Gradient of phi written to grad_phi.vtk" << std::endl;
    } else {
        std::cerr << "Error: could not open grad_phi.vtk for writing" << std::endl;
    }

    delete [] h_pts;
    hipFree(pts);
    hipFree(area);
    hipFree(phi);
    delete [] h_phi;
    
    // cudaFree(grad_phi);
    delete [] h_grad_phi;
    // cudaFree(grad_phi_ref);
    delete [] h_grad_phi_ref;
    
    return 0;
}