#include "hip/hip_runtime.h"
#include "LaplaceHeat.h"
#include <iostream>
#include <fstream>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>

namespace LaplaceHeatNS {

    // Functor to square the elements
    struct square {
        __device__ double operator()(double a) {
            return a * a;
        }
    };

// Kernel function for initialization - No tiling or shared memory
__global__ void initialize_const(double *T, double val, int nx, int ny) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < (nx * ny)) 
        T[idx] = val ;
    
}

// Kernel function for initialization - No tiling or shared memory
__global__ void initialize_ref(double *T, int nx, int ny, double dx, double dy) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < (nx * ny)) {
        int row = idx % nx;
        int col = idx / nx;
        double y = (0.5 + col) * dy;
        double x = (0.5 + row) * dx;
        T[(col * nx) + row] = 300.0 + x*x + (y*y*y)/ 27.0;
    }
    
}
// Kernel function for update - No tiling or shared memory
__global__ void update(double *T, double *deltaT, int nx, int ny) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < (nx * ny)) 
        T[idx] += deltaT[idx];

}

// Kernel function for calculation of Jacobian and Residual - No tiling or shared memory
__global__ void compute_r_j(double *T, double *J, double *R, int nx, int ny, double dx, double dy, double kc) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if ((row < nx) && (col < ny)) {
        
        double y = (0.5 + col) * dy;
        double x = (0.5 + row) * dx;
        int idx_r = (col * nx) + row;
        int idx_j = idx_r * 5;

        double jij = -4.0;
        double jip1j = 1.0;
        double jim1j = 1.0;
        double jijp1 = 1.0;
        double jijm1 = 1.0;

        double tip1j = 0.0;
        double tim1j = 0.0;
        double tijp1 = 0.0;
        double tijm1 = 0.0;

        double radd = 0.0;
        if (row == 0) {
            jij -= 2.0;
            jip1j += 0.3333333333333333 ;
            jim1j -= 1.0;
            tip1j = T[idx_r + 1];
            double t_bc_left = 300.0 + (y*y*y/27.0);
            radd += kc * 8.0 * t_bc_left / 3.0 ;
        } else if (row == (nx - 1)) {
            jij -= 2.0;
            jim1j += 0.3333333333333333;
            jip1j -= 1.0;
            tim1j = T[idx_r - 1];
            double t_bc_right = 300.0 + 1.0 + (y*y*y/27.0);
            radd += kc * 8.0 * t_bc_right / 3.0;
        } else {
            tip1j = T[idx_r + 1];
            tim1j = T[idx_r - 1];
        }

        if (col == 0) {
            jij -= 2.0;
            jijp1 += 0.3333333333333333;
            jijm1 -= 1.0;
            tijp1 = T[idx_r + nx];
            double t_bc_bot = 300.0 + (x*x);
            radd += kc * 8.0 * t_bc_bot / 3.0;
        } else if (col == (ny - 1)) {
            jij -= 2.0;
            jijm1 += 0.3333333333333333;
            jijp1 -= 1.0;
            tijm1 = T[idx_r - nx];
            double t_bc_top = 300.0 + 1.0 + (x*x);
            radd += kc * 8.0 * t_bc_top / 3.0;
        } else {
            tijm1 = T[idx_r - nx];
            tijp1 = T[idx_r + nx];
        }

        // Write to residual
        double tmp = kc * ( jijm1 * tijm1 + jijp1 * tijp1 + jim1j * tim1j + jip1j * tip1j + jij * T[idx_r] - (2.0 + 2.0 * y / 9.0) * dx * dy) + radd;

        // if (std::abs(tmp/(dx * dy * kc)) > 20.0) {
        //     printf("Row, Col is %d, %d - x,y = %f, %f, Residuals - %f, %f, J - (j-1) %f, (j+1) %f, (i-1) %f, (i+1) %f, (ij) %f, T - (j-1) %f, (j+1) %f, (i-1) %f, (i+1) %f, (ij) %f \n", row, col, x, y, 2.0 - 2.0 * y / 9.0, tmp / (dx * dy * kc), jijm1, jijp1, jim1j, jip1j, jij, tijm1, tijp1, tim1j, tip1j, T[idx_r]);
        // }

        R[idx_r] = -tmp;

        // Write to the Jacobian
        J[idx_j] = jij * kc; //i,j
        J[idx_j + 1] = jim1j * kc; //i-1,j
        J[idx_j + 2] = jip1j * kc; //i+1,j
        J[idx_j + 3] = jijm1 * kc; //i,j-1
        J[idx_j + 4] = jijp1 * kc; //i,j+1
    }
}

// Kernel function for calculation of Residual - No tiling or shared memory
__global__ void compute_r(double *T, double * J, double *R, int nx, int ny, double dx, double dy, double kc) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if ((row < nx) && (col < ny)) {
        
        double y = (0.5 + col) * dy;
        double x = (0.5 + row) * dx;
        int idx_r = (col * nx) + row;
        int idx_j = idx_r * 5;

        double jij = J[idx_j];
        double jim1j = J[idx_j + 1];
        double jip1j = J[idx_j + 2];
        double jijm1 = J[idx_j + 3];
        double jijp1 = J[idx_j + 4];

        double tip1j = 0.0;
        double tim1j = 0.0;
        double tijp1 = 0.0;
        double tijm1 = 0.0;

        double radd = 0.0;
        if (row == 0) {
            tip1j = T[idx_r + 1];
            double t_bc_left = 300.0 + (y*y*y/27.0);
            radd += kc * 8.0 * t_bc_left / 3.0 ;
        } else if (row == (nx - 1)) {
            tim1j = T[idx_r - 1];
            double t_bc_right = 300.0 + 1.0 + (y*y*y/27.0);
            radd += kc * 8.0 * t_bc_right / 3.0;
        } else {
            tip1j = T[idx_r + 1];
            tim1j = T[idx_r - 1];
        }

        if (col == 0) {
            tijp1 = T[idx_r + nx];
            double t_bc_bot = 300.0 + (x*x);
            radd += kc * 8.0 * t_bc_bot / 3.0;
        } else if (col == (ny - 1)) {            
            tijm1 = T[idx_r - nx];
            double t_bc_top = 300.0 + 1.0 + (x*x);
            radd += kc * 8.0 * t_bc_top / 3.0;
        } else {
            tijm1 = T[idx_r - nx];
            tijp1 = T[idx_r + nx];
        }

        // Write to residual
        R[idx_r] = -kc * ( jijm1 * tijm1 + jijp1 * tijp1 + jim1j * tim1j + jip1j * tip1j + jij * T[idx_r] - (2.0 + 2.0 * y / 9.0) * dx * dy) - radd;
    }
}


// Kernel to compute matrix vector product of the linear system of equations J * v . 
__global__ void compute_matvec(double * v, double * J, double * result, int nx, int ny) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int idx_r = (j * nx) + i;
    int idx_j = idx_r * 5;

    if ( (i < nx) && (j < ny)) {

        double jij = J[idx_j];
        double jim1j = J[idx_j + 1];
        double jip1j = J[idx_j + 2];
        double jijm1 = J[idx_j + 3];
        double jijp1 = J[idx_j + 4];

        double vip1j = 0.0;
        double vim1j = 0.0;
        double vijp1 = 0.0;
        double vijm1 = 0.0;

        if ( i == 0) {
            vip1j = v[idx_r + 1];
        } else if ( i == (nx - 1)) {
            vim1j = v[idx_r - 1];
        } else {
            vip1j = v[idx_r + 1];
            vim1j = v[idx_r - 1];
        }

        if ( j == 0) {
            vijp1 = v[idx_r + nx];
        } else if ( j == (ny - 1)) {
            vijm1 = v[idx_r - nx];
        } else {
            vijm1 = v[idx_r - nx];
            vijp1 = v[idx_r + nx];
        }

        result[idx_r] = jim1j * vim1j + jip1j * vip1j + jijm1 * vijm1 + jijp1 * vijp1 + jij * v[idx_r];
    }
}

    LaplaceHeat::LaplaceHeat(int nx_inp, int ny_inp, double kc_inp, std::string solver_type) {

        nx = nx_inp;
        ny = ny_inp;
        kc = kc_inp;
        hipMalloc(&T, nx * ny * sizeof(double));
        hipMalloc(&deltaT, nx * ny * sizeof(double));
        hipMalloc(&J, nx * ny * 5 * sizeof(double));
        hipMalloc(&nlr, nx * ny * sizeof(double));

        double dx = 1.0 / double(nx);
        double dy = 3.0 / double(ny);

        grid_size = dim3(nx, ny);
        block_size = dim3(32, 32);

        grid_size_1d = dim3( ceil (nx * ny / 1024.0) );       

        if (solver_type == "Jacobi") {
            solver = new JacobiNS::Jacobi(nx, ny, J, T, deltaT, nlr);
        } else if (solver_type == "ADI" ) {
            solver = new ADINS::ADI(nx, ny, J, T, deltaT, nlr);
        } else {
            std::cout << "Invalid solver type. Availabl solvers are Jacobi and ADI. " << std::endl;
            exit(1);
        }
        
    }

    LaplaceHeat::~LaplaceHeat() {

        // Free memory
        hipFree(T);
        hipFree(deltaT);
        hipFree(J);
        hipFree(nlr);
    }


    __host__ void LaplaceHeat::initialize_const(double val) {
        LaplaceHeatNS::initialize_const<<<grid_size_1d, block_size_1d>>>(T, val, nx, ny);
        hipDeviceSynchronize();
    }

    __host__ void LaplaceHeat::initialize_ref() {
        LaplaceHeatNS::initialize_ref<<<grid_size_1d, block_size_1d>>>(T, nx, ny, dx, dy);
        hipDeviceSynchronize();
    }

    __host__ void LaplaceHeat::update() {
        LaplaceHeatNS::update<<<grid_size_1d, block_size_1d>>>(T, deltaT, nx, ny);
        hipDeviceSynchronize();
    }

    __host__ double LaplaceHeat::compute_r_j() {
        LaplaceHeatNS::compute_r_j<<<grid_size, block_size>>>(T, J, nlr, nx, ny, dx, dy, kc);        
        hipDeviceSynchronize();
        thrust::device_ptr<double> t_nlr(nlr);
        return std::sqrt(thrust::transform_reduce(t_nlr, t_nlr + nx * ny, square(), 0.0, thrust::plus<double>()));        
    }

    __host__ double LaplaceHeat::compute_r() {
        LaplaceHeatNS::compute_r<<<grid_size, block_size>>>(T, J, nlr, nx, ny, dx, dy, kc);
        hipDeviceSynchronize();
        thrust::device_ptr<double> t_nlr(nlr);
        return std::sqrt(thrust::transform_reduce(t_nlr, t_nlr + nx * ny, square(), 0.0, thrust::plus<double>()));              
    }

    __host__ void LaplaceHeat::compute_matvec(double * v, double * result) {
        LaplaceHeatNS::compute_matvec<<<grid_size, block_size>>>(v, J, result, nx, ny);
        hipDeviceSynchronize();
    }

    __host__ void LaplaceHeat::solve(int nsteps) {
        for (int j = 0; j < nsteps; j++) {
            solver->solve_step();
    }

}

}


int main() {

    // std::ofstream resid_file_jacobi("jacobi_resid.txt");
    // resid_file_jacobi << "Iter, Residual" << std::endl;
    // LaplaceHeatNS::LaplaceHeat * ljacobi = new LaplaceHeatNS::LaplaceHeat(128, 384, 0.01, "Jacobi");
    // ljacobi->initialize_const(300.0);
    double * resid = new double[80];
    // for (int i = 0; i < 80; i++) {
    //     resid[i] = ljacobi->compute_r_j();
    //     resid_file_jacobi << i << ", " << resid[i] << std::endl;
    //     ljacobi->solve(1000); // Loops of Jacobi
    //     ljacobi->update();
    // }
    // resid_file_jacobi.close();
    // delete ljacobi;

    std::ofstream resid_file_adi("adi_resid.txt");
    resid_file_adi << "Iter, Residual" << std::endl;
    LaplaceHeatNS::LaplaceHeat * ladi = new LaplaceHeatNS::LaplaceHeat(128, 384, 0.01, "ADI");
    ladi->initialize_const(300.0);
    for (int i = 0; i < 80; i++) {
        resid[i] = ladi->compute_r_j();
        resid_file_adi << "Iter = " << i << "resid = " << resid[i] << std::endl;
        ladi->solve(100); // Loops of ADI
        ladi->update();
    }
    resid_file_adi.close();
    delete ladi;

    return 0;
    
}

