#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <thrust/reduce.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>

#define TILE_SIZE 32
#define TILE_SIZE_ADI 1

// Kernel function for initialization - No tiling or shared memory
__global__ void initialize(double *T, int nx, int ny, double dx, double dy);

// Kernel function for initialization - No tiling or shared memory
__global__ void initialize_ref(double *T, int nx, int ny, double dx, double dy);

// Kernel function for update - No tiling or shared memory
__global__ void update(double *T, double *deltaT, int nx, int ny, double dx, double dy);

// Kernel function for calculation of Jacobian and Residual - No tiling or shared memory
__global__ void compute_r_j(double *T, double *J, double *R, int nx, int ny, double dx, double dy, double kc);

// Kernel function for calculation of Residual - No tiling or shared memory
__global__ void compute_r(double *T, double * J, double *R, int nx, int ny, double dx, double dy, double kc) ;

// Kernel function for Thomas solves in the X direction - part of ADI 
__global__ void adi_x(double *T, double *J, double *R, int nx, int ny);

// Kernel function for Thomas solves in the Y direction - part of ADI 
__global__ void adi_y(double *T, double *J, double *R, int nx, int ny);

// Functor to square the elements
struct square {
    __device__ double operator()(double a) {
        return a * a;
    }
};

// Kernel function to initialize a given field to zero
__global__ void initialize_zero(double * T, int nx, int ny) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = (j * nx) + i;

    if ( (i < nx) && (j < ny))
        T[idx] = 0.0;
}

// Kernel to compute linear residual of the linear system of equations J * deltaT = rhs. 
// Write linear residual to new array R. If you want the rhs overwritten, pass the same pointers for rhs and R
__global__ void compute_lin_resid(double * deltaT, double * J, double * rhs, double * R, int nx, int ny) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int idx_r = (j * nx) + i;
    int idx_j = idx_r * 5;

    if ( (i < nx) && (j < ny)) {

        double jij = J[idx_j];
        double jim1j = J[idx_j + 1];
        double jip1j = J[idx_j + 2];
        double jijm1 = J[idx_j + 3];
        double jijp1 = J[idx_j + 4];

        double tip1j = 0.0;
        double tim1j = 0.0;
        double tijp1 = 0.0;
        double tijm1 = 0.0;

        if ( i == 0) {
            tip1j = deltaT[idx_r + 1];
        } else if ( i == (nx - 1)) {
            tim1j = deltaT[idx_r - 1];
        } else {
            tip1j = deltaT[idx_r + 1];
            tim1j = deltaT[idx_r - 1];
        }

        if ( j == 0) {
            tijp1 = deltaT[idx_r + nx];
        } else if ( j == (ny - 1)) {
            tijm1 = deltaT[idx_r - nx];
        } else {
            tijm1 = deltaT[idx_r - nx];
            tijp1 = deltaT[idx_r + nx];
        }

        // Write to residual

        if (std::abs(jij) < 1e-5) {
            printf("nx = %d, ny = %d, i = %d, j = %d, R = %e, jim1j = %e, jip1j = %e, jijm1 = %e, jijp1 = %e, jij = %e \n", nx, ny, i, j, R[idx_r], jim1j, jip1j, jijm1, jijp1, jij);
        }
        R[idx_r] = rhs[idx_r] - jim1j * tim1j - jip1j * tip1j - jijm1 * tijm1 - jijp1 * tijp1 - jij * deltaT[idx_r];
    }
}

// Restrict residual by one level. Expected that nxc = nxf/2 and nyc = nyf/2
// Expected to run on a grid and block that represents the coarse mesh
__global__ void restrict_resid(double * rc, double * rf, int nxc, int nyc, int nxf, int nyf) {

    int ic = blockIdx.x * blockDim.x + threadIdx.x;
    int jc = blockIdx.y * blockDim.y + threadIdx.y;

    // Each cell in the coarse mesh (ic, jc) is a sum of the 4 cells corresponding to (2 * ic, 2 * jc), (2 * ic + 1, 2 * jc), (2 * ic, 2 * jc + 1), (2 * ic + 1, 2 * jc + 1)
    int idx_rc = (jc * nxc) + ic;
    int idx_rf1 = (2 * jc * nxf) + (2 * ic);
    int idx_rf2 = (2 * jc * nxf) + (2 * ic + 1);
    int idx_rf3 = (2 * jc + 1) * nxf + (2 * ic);
    int idx_rf4 = (2 * jc + 1) * nxf + (2 * ic + 1);

    if ( (ic < nxc) && (jc < nyc) ) 
        rc[idx_rc] = (rf[idx_rf1] + rf[idx_rf2] + rf[idx_rf3] + rf[idx_rf4]);//std::sqrt(2.0);

}

// Prolongate error by one level. Expected that nxc = nxf/2 and nyc = nyf/2
// Expected to run on a grid and block that represents the coarse mesh
__global__ void prolongate_error(double * deltaTc, double * deltaTf, int nxc, int nyc, int nxf, int nyf) {

    int ic = blockIdx.x * blockDim.x + threadIdx.x;
    int jc = blockIdx.y * blockDim.y + threadIdx.y;

    // Each cell in the coarse mesh (ic, jc) is a sum of the 4 cells corresponding to (2 * ic, 2 * jc), (2 * ic + 1, 2 * jc), (2 * ic, 2 * jc + 1), (2 * ic + 1, 2 * jc + 1)
    int idx_rc = (jc * nxc) + ic;
    int idx_rf1 = (2 * jc * nxf) + (2 * ic);
    int idx_rf2 = (2 * jc * nxf) + (2 * ic + 1);
    int idx_rf3 = (2 * jc + 1) * nxf + (2 * ic);
    int idx_rf4 = (2 * jc + 1) * nxf + (2 * ic + 1);

    if ( (ic < nxc) && (jc < nyc) ) {

        //printf("Prolongating (i,j), idx_rc = %d, %d, %d deltaT = %e, idx_rf1 = %d, idx_rf2 = %d, idx_rf3 = %d, idx_rf4 = %d \n", ic, jc, idx_rc, deltaTc[idx_rc], idx_rf1, idx_rf2, idx_rf3, idx_rf4);
        deltaTf[idx_rf1] += 0.9*deltaTc[idx_rc];//std::sqrt(2.0);
        deltaTf[idx_rf2] += 0.9*deltaTc[idx_rc];//std::sqrt(2.0);
        deltaTf[idx_rf3] += 0.9*deltaTc[idx_rc];//std::sqrt(2.0);
        deltaTf[idx_rf4] += 0.9*deltaTc[idx_rc];//std::sqrt(2.0);
        
    }    

}


// Create Jacobian matrix at the coarser level by using the finer level. Expected that nxc = nxf/2 and nyc = nyf/2
// Expected to run on a grid and block that represents the coarse mesh
__global__ void restrict_j(double * jc, double * jf, int nxc, int nyc, int nxf, int nyf)  {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Each cell in the coarse mesh (ic, jc) is a sum of the 4 cells corresponding to (2 * ic, 2 * jc), (2 * ic + 1, 2 * jc), (2 * ic, 2 * jc + 1), (2 * ic + 1, 2 * jc + 1)
    int idx_jc = ((j * nxc) + i) * 5;
    int idx_jf1 = ((2 * j * nxf) + (2 * i)) * 5;
    int idx_jf2 = ((2 * j * nxf) + (2 * i + 1)) * 5;
    int idx_jf3 = ((2 * j + 1) * nxf + (2 * i)) * 5;
    int idx_jf4 = ((2 * j + 1) * nxf + (2 * i + 1)) * 5;

    if ( (i < nxc) && (j < nyc) ) {

        // Diagonals and Interlinks of the 4 cells
        jc[idx_jc] = (jf[idx_jf1] + jf[idx_jf2] + jf[idx_jf3] + jf[idx_jf4]    + jf[idx_jf1+2] + jf[idx_jf1+4]    + jf[idx_jf2+1] + jf[idx_jf2+4]    + jf[idx_jf3+3] + jf[idx_jf3+2]    + jf[idx_jf4+1] + jf[idx_jf4+3]);
        jc[idx_jc+1] = (jf[idx_jf1+1] + jf[idx_jf3+1]);
        jc[idx_jc+2] = (jf[idx_jf2+2] + jf[idx_jf4+2]);
        jc[idx_jc+3] = (jf[idx_jf1+3] + jf[idx_jf2+3]);
        jc[idx_jc+4] = (jf[idx_jf3+4] + jf[idx_jf4+4]);
        
    }

}

// Kernel function for Gauss-Seidel smoother - No tiling or shared memory
__global__ void jacobi(double *deltaT, double * deltaT1, double *J, double *R, int nx, int ny) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (( i < nx) && (j < ny)) {
        
        int idx_r = (j * nx) + i;
        int idx_j = idx_r * 5;

        double jij = J[idx_j];
        double jim1j = J[idx_j + 1];
        double jip1j = J[idx_j + 2];
        double jijm1 = J[idx_j + 3];
        double jijp1 = J[idx_j + 4];

        double tip1j = 0.0;
        double tim1j = 0.0;
        double tijp1 = 0.0;
        double tijm1 = 0.0;

        if (i == 0) {
            tip1j = deltaT[idx_r + 1];
        } else if (i == (nx - 1)) {
            tim1j = deltaT[idx_r - 1];
        } else {
            tip1j = deltaT[idx_r + 1];
            tim1j = deltaT[idx_r - 1];
        }

        if (j == 0) {
            tijp1 = deltaT[idx_r + nx];
        } else if (j == (ny - 1)) {
            tijm1 = deltaT[idx_r - nx];
        } else {
            tijm1 = deltaT[idx_r - nx];
            tijp1 = deltaT[idx_r + nx];
        }

        deltaT1[idx_r] = (R[idx_r] - jim1j * tim1j - jip1j * tip1j - jijm1 * tijm1 - jijp1 * tijp1) / jij;

        if (std::abs(jij) < 1e-8)
            printf("nx = %d, ny = %d, i = %d, j = %d, deltaT = %e, R = %e, jim1j = %e, jip1j = %e, jijm1 = %e, jijp1 = %e, jij = %e \n", nx, ny, i, j, deltaT[idx_r], R[idx_r], jim1j, jip1j, jijm1, jijp1, jij);


        // if (std::isinf(deltaT[idx_r]) || std::isnan(deltaT[idx_r]))
    }
}


int main() {

    // Finest level problem size
    int nx_f = 128*16;
    int ny_f = 384*16;

    // Need resolution only on the finest grid to assemble the equations
    double dx = 1.0 / double(nx_f);
    double dy = 3.0 / double(ny_f);

    double kc = 0.01;

    // Number of levels in multigrid - each refined in all directions by a factor of 2
    int nlevels = 9; 
    std::vector<int> nx(nlevels);
    std::vector<int> ny(nlevels);
    for (int i = 0; i < nlevels; i++) {
        nx[i] = nx_f / (1 << i);
        ny[i] = ny_f / (1 << i);
        std::cout << "ilevel = " << i << ", nx = " << nx[i] << ", ny = " << ny[i] << std::endl;
    }

    // Fields for temperature and non-linear residual are required only at the finest level
    double * T;
    hipMalloc(&T, nx_f * ny_f * sizeof(double));
    double * nlr;
    hipMalloc(&nlr, nx_f * ny_f * sizeof(double));

    std::vector<double*> deltaT(nlevels), deltaT1(nlevels), J(nlevels), R(nlevels), Rlin(nlevels);
    for (int i = 0; i < nlevels; i++) {
        hipMalloc(&deltaT[i], nx[i] * ny[i] * sizeof(double));
        hipMalloc(&deltaT1[i], nx[i] * ny[i] * sizeof(double));
        hipMalloc(&J[i], nx[i] * ny[i] * 5 * sizeof(double));
        hipMalloc(&R[i], nx[i] * ny[i] * sizeof(double));
        hipMalloc(&Rlin[i], nx[i] * ny[i] * sizeof(double));
    }

    // Grid and block size
    std::vector<dim3> grid_size;
    for (int ilevel = 0; ilevel < nlevels; ilevel++) 
        grid_size.push_back(dim3(ceil(nx[ilevel] / (double)TILE_SIZE), ceil(ny[ilevel] / (double)TILE_SIZE), 1));
    // Keep block size same for all grids for now
    dim3 block_size(TILE_SIZE, TILE_SIZE, 1);

    initialize<<<grid_size[0], block_size>>>(T, nx[0], ny[0], dx, dy);
    hipDeviceSynchronize();

    compute_r_j<<<grid_size[0], block_size>>>(T, J[0], nlr, nx[0], ny[0], dx, dy, kc);
    hipDeviceSynchronize();
    double glob_resid = 0.0;
    thrust::device_ptr<double> t_nlr(nlr);
    glob_resid = std::sqrt(thrust::transform_reduce(t_nlr, t_nlr + nx[0] * ny[0], square(), 0.0, thrust::plus<double>()));
    std::cout << "Starting residual with const 300.0 field = " << glob_resid << std::endl;
    
    // Compute the Jacobian matrix at the coarser levels 
    for (int ilevel = 1; ilevel < nlevels; ilevel++) {
        restrict_j<<<grid_size[ilevel], block_size>>>(J[ilevel], J[ilevel-1], nx[ilevel], ny[ilevel], nx[ilevel-1], ny[ilevel-1]);
        hipDeviceSynchronize();
    }

    // Write 1 V-cycle of multigrid

    for (int iloop = 0; iloop < 100; iloop++) {    
    // Downstroke of V-cycle

    // Initialize deltaT at all levels to zero
    for (int ilevel = 0; ilevel < nlevels; ilevel++) {
        initialize_zero<<<grid_size[ilevel], block_size>>>(deltaT[ilevel], nx[ilevel], ny[ilevel]);
        hipDeviceSynchronize();
        initialize_zero<<<grid_size[ilevel], block_size>>>(deltaT1[ilevel], nx[ilevel], ny[ilevel]);
        hipDeviceSynchronize();
    }

    // Do some smoothing on the finest level first
    for (int ismooth = 0; ismooth < 10; ismooth++) {
        jacobi<<<grid_size[0], block_size>>>(deltaT[0], deltaT1[0], J[0], nlr, nx[0], ny[0]);
        hipDeviceSynchronize();
        jacobi<<<grid_size[0], block_size>>>(deltaT1[0], deltaT[0], J[0], nlr, nx[0], ny[0]);
        hipDeviceSynchronize();
    }

    // // Compute the residual of the linear system of equations at this level
    compute_lin_resid<<<grid_size[0], block_size>>>(deltaT[0], J[0], nlr, Rlin[0], nx[0], ny[0]);
    hipDeviceSynchronize();

    for (int ilevel = 1; ilevel < nlevels-1; ilevel++) {
        // Restrict the residual of the linear system
        restrict_resid<<<grid_size[ilevel], block_size>>>(R[ilevel], Rlin[ilevel-1], nx[ilevel], ny[ilevel], nx[ilevel-1], ny[ilevel-1]);
        hipDeviceSynchronize();
        
        // Perform some smoothing at this level to get the error
        for (int ismooth = 0; ismooth < 10; ismooth++) {
            jacobi<<<grid_size[ilevel], block_size>>>(deltaT[ilevel], deltaT1[ilevel], J[ilevel], R[ilevel], nx[ilevel], ny[ilevel]);
            hipDeviceSynchronize();
            jacobi<<<grid_size[ilevel], block_size>>>(deltaT1[ilevel], deltaT[ilevel], J[ilevel], R[ilevel], nx[ilevel], ny[ilevel]);
            hipDeviceSynchronize();            
        }

        // Compute the residual of the linear system of equations at this level.
        compute_lin_resid<<<grid_size[ilevel], block_size>>>(deltaT[ilevel], J[ilevel], R[ilevel], Rlin[ilevel], nx[ilevel], ny[ilevel]);
        hipDeviceSynchronize();

    }

    // Restrict the residual of the linear system to coarsest level
    restrict_resid<<<grid_size[nlevels-1], block_size>>>(R[nlevels-1], Rlin[nlevels-2], nx[nlevels-1], ny[nlevels-1], nx[nlevels-2], ny[nlevels-2]);
    hipDeviceSynchronize();

    // Do bottom level solve with ADI 
    dim3 grid_size_adix(ceil(ny[nlevels-1] / (double)TILE_SIZE_ADI), 1, 1);
    dim3 block_size_adi(TILE_SIZE_ADI, 1,1);
    dim3 grid_size_adiy(ceil(nx[nlevels-1] / (double)TILE_SIZE_ADI), 1, 1);

    for (int ismooth = 0; ismooth < 100; ismooth++) {

        compute_lin_resid<<<grid_size[nlevels-1], block_size>>>(deltaT[nlevels-1], J[nlevels-1], R[nlevels-1], Rlin[nlevels-1], nx[nlevels-1], ny[nlevels-1]);        
        hipDeviceSynchronize();

        // jacobi<<<grid_size[nlevels-1], block_size>>>(deltaT[nlevels-1], deltaT1[nlevels-1], J[nlevels-1], R[nlevels-1], nx[nlevels-1], ny[nlevels-1]);
        // hipDeviceSynchronize();  
        // jacobi<<<grid_size[nlevels-1], block_size>>>(deltaT1[nlevels-1], deltaT[nlevels-1], J[nlevels-1], R[nlevels-1], nx[nlevels-1], ny[nlevels-1]);
        // hipDeviceSynchronize();                      
        adi_x<<<grid_size_adix, block_size_adi>>>(deltaT[nlevels-1], J[nlevels-1], R[nlevels-1], nx[nlevels-1], ny[nlevels-1]);
        hipDeviceSynchronize();
        adi_y<<<grid_size_adiy, block_size_adi>>>(deltaT[nlevels-1], J[nlevels-1], R[nlevels-1], nx[nlevels-1], ny[nlevels-1]);
        hipDeviceSynchronize();
    }


    // Upstroke of V-cycle - This should end on the finest level (ilevel = 0)
    for (int ilevel = nlevels - 2; ilevel > 0; ilevel--) {
        // Prolongate the error
        prolongate_error<<<grid_size[ilevel+1], block_size>>>(deltaT[ilevel+1], deltaT[ilevel], nx[ilevel+1], ny[ilevel+1], nx[ilevel], ny[ilevel]);
        hipDeviceSynchronize();

        // Do some more smoothing at this level to reduce the error
        for (int ismooth = 0; ismooth < 10; ismooth++) {     
            jacobi<<<grid_size[ilevel], block_size>>>(deltaT[ilevel], deltaT1[ilevel], J[ilevel], R[ilevel], nx[ilevel], ny[ilevel]);
            hipDeviceSynchronize();
            jacobi<<<grid_size[ilevel], block_size>>>(deltaT1[ilevel], deltaT[ilevel], J[ilevel], R[ilevel], nx[ilevel], ny[ilevel]);
            hipDeviceSynchronize();
        }
    }

    prolongate_error<<<grid_size[1], block_size>>>(deltaT[1], deltaT[0], nx[1], ny[1], nx[0], ny[0]);
    hipDeviceSynchronize();

    for (int ismooth=0; ismooth < 10; ismooth++) {
        jacobi<<<grid_size[0], block_size>>>(deltaT[0], deltaT1[0], J[0], nlr, nx[0], ny[0]);
        hipDeviceSynchronize();
        jacobi<<<grid_size[0], block_size>>>(deltaT1[0], deltaT[0], J[0], nlr, nx[0], ny[0]);
        hipDeviceSynchronize();        
    }

    update<<<grid_size[0], block_size>>>(T, deltaT[0], nx[0], ny[0], dx, dy);
    hipDeviceSynchronize();

    compute_r_j<<<grid_size[0], block_size>>>(T, J[0], nlr, nx[0], ny[0], dx, dy, kc);
    hipDeviceSynchronize();
    glob_resid = std::sqrt(thrust::transform_reduce(t_nlr, t_nlr + nx[0] * ny[0], square(), 0.0, thrust::plus<double>()));
    std::cout << "Loop = " << iloop << ", Ending residual = " << glob_resid << std::endl;

    }


    double *h_R = new double[nx_f * ny_f];
    hipMemcpy(h_R, nlr, nx_f * ny_f * sizeof(double), hipMemcpyDeviceToHost);

    // Write h_R to a file 
    std::ofstream outfile("residual.txt");
    for (int j = 0; j < ny_f; ++j) {
        for (int i = 0; i < nx_f; ++i) {
            outfile << h_R[j * nx_f + i] << " ";
        }
        outfile << std::endl;
    }
    outfile.close();
    delete[] h_R;    

    double *h_T = new double[nx_f * ny_f];
    hipMemcpy(h_T, T, nx_f * ny_f * sizeof(double), hipMemcpyDeviceToHost);

    // Write h_T to a file
    std::ofstream tfile("temperature_output.txt");
    for (int j = 0; j < ny_f; ++j) {
        for (int i = 0; i < nx_f; ++i) {
            tfile << h_T[j * nx_f + i] << " ";
        }
        tfile << std::endl;
    }
    tfile.close();
    delete[] h_T;    

    return 0;
}