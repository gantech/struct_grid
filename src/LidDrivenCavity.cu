#include "hip/hip_runtime.h"
#include <iostream>
#include "LidDrivenCavity.h"

namespace LidDridenCavityNS {

    LidDrivenCavity::LidDridenCavity(int nx_inp, int ny_inp, double nu_inp) {

        nx = nx_inp;
        ny = ny_inp;
        nu = nu_inp;
        
        grid_size = dim3(nx, ny);
        block_size = dim3(32, 32);

        grid_size_1d = dim3( ceil (nx * ny / 1024.0) );       

        hipMalloc(&umom, nx * ny * sizeof(double));
        hipMalloc(&vmom, nx * ny * sizeof(double));
        hipMalloc(&pres, nx * ny * sizeof(double));
        hipMalloc(&deltaU, nx * ny * sizeof(double));
        hipMalloc(&deltaV, nx * ny * sizeof(double));
        hipMalloc(&deltaP, nx * ny * sizeof(double));
        hipMalloc(&Jmom, nx * ny * 5 * sizeof(double));
        hipMalloc(&Jcont, nx * ny * 5 * sizeof(double));
        hipMalloc(&u_nlr, nx * ny * sizeof(double));
        hipMalloc(&v_nlr, nx * ny * sizeof(double));
        hipMalloc(&p_nlr, nx * ny * sizeof(double));

        std::cout << "Allocated " <<  19 * nx * ny * sizeof(double) / std::double(1 << 30) << " GB of memory" << std::endl;
        t_unlr = thrust::device_ptr<double>(u_nlr);
        t_vnlr = thrust::device_ptr<double>(v_nlr);
        t_cont_nlr = thrust::device_ptr<double>(cont_nlr);

    }

    LidDridenCavity::~LidDrivenCavity() {

        hipFree(umom);
        hipFree(vmom);
        hipFree(pres);
        hipFree(deltaU);
        hipFree(deltaV);
        hipFree(deltaP);
        hipFree(Jmom);
        hipFree(Jcont);
        hipFree(u_nlr);
        hipFree(v_nlr);
        hipFree(p_nlr);

    }
    
}