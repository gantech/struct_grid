#include "hip/hip_runtime.h"
#include <iostream>
#include "LidDrivenCavity.h"

namespace LidDrivenCavityNS {

    LidDrivenCavity::LidDrivenCavity(int nx_inp, int ny_inp, double nu_inp) {

        nx = nx_inp;
        ny = ny_inp;
        dx = 1.0 / nx;
        dy = 3.0 / ny;
        nu = nu_inp;
        
        grid_size = dim3(nx, ny);
        block_size = dim3(32, 32);

        grid_size_1d = dim3( ceil (nx * ny / 1024.0) );       

        hipMalloc(&umom, nx * ny * sizeof(double));
        hipMalloc(&vmom, nx * ny * sizeof(double));
        hipMalloc(&pres, nx * ny * sizeof(double));
        hipMalloc(&deltaU, nx * ny * sizeof(double));
        hipMalloc(&deltaV, nx * ny * sizeof(double));
        hipMalloc(&deltaP, nx * ny * sizeof(double));
        hipMalloc(&Jmom, nx * ny * 5 * sizeof(double));
        hipMalloc(&Jcont, nx * ny * 5 * sizeof(double));
        hipMalloc(&u_nlr, nx * ny * sizeof(double));
        hipMalloc(&v_nlr, nx * ny * sizeof(double));
        hipMalloc(&cont_nlr, nx * ny * sizeof(double));

        std::cout << "Allocated " <<  19 * nx * ny * sizeof(double) / double(1 << 30) << " GB of memory" << std::endl;
        t_unlr = thrust::device_ptr<double>(u_nlr);
        t_vnlr = thrust::device_ptr<double>(v_nlr);
        t_cont_nlr = thrust::device_ptr<double>(cont_nlr);

    }

    LidDrivenCavity::~LidDrivenCavity() {

        hipFree(umom);
        hipFree(vmom);
        hipFree(pres);
        hipFree(deltaU);
        hipFree(deltaV);
        hipFree(deltaP);
        hipFree(Jmom);
        hipFree(Jcont);
        hipFree(u_nlr);
        hipFree(v_nlr);
        hipFree(cont_nlr);

    }
    
}

int main() {

    LidDrivenCavityNS::LidDrivenCavity * lcav = new LidDrivenCavityNS::LidDrivenCavity(128, 384, 0.001);

    delete lcav;
    return 0;
}